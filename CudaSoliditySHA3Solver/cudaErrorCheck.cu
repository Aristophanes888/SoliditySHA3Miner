#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall(err)		__cudaSafeCall(err, __FILE__, __LINE__, deviceID)
#define CudaSyncAndCheckError()	__cudaSyncAndCheckError(__FILE__, __LINE__, deviceID)

__host__ inline std::string __cudaSafeCall(hipError_t err, const char *file, const int line, const int deviceID)
{
#ifdef CUDA_ERROR_CHECK
	if (hipSuccess != err)
		return "CUDA device ID [" + std::to_string(deviceID) + "] encountered an error: " + hipGetErrorString(err);
	else
#endif //CUDA_ERROR_CHECK
		return "";
}

__host__ inline std::string __cudaSyncAndCheckError(const char *file, const int line, const int deviceID)
{
	hipError_t response{ hipSuccess };
	std::string cudaErrors{ "" };

#ifdef CUDA_ERROR_CHECK
	response = hipGetLastError();
	if (response != hipSuccess)
	{
		while (response != hipSuccess)
		{
			if (!cudaErrors.empty()) cudaErrors += " <- ";
			cudaErrors += hipGetErrorString(response);
			response = hipGetLastError();
		}
		return "CUDA device ID [" + std::to_string(deviceID) + "] encountered an error: " + cudaErrors;
	}
#endif //CUDA_ERROR_CHECK

	response = hipDeviceSynchronize();
	if (response != hipSuccess)
	{
		while (response != hipSuccess)
		{
			if (!cudaErrors.empty()) cudaErrors += " <- ";
			cudaErrors += hipGetErrorString(response);
			response = hipGetLastError();
		}
		return "CUDA device ID [" + std::to_string(deviceID) + "] encountered an error: " + cudaErrors;
	}
	return "";
}
